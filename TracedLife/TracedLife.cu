#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <gtc/type_ptr.hpp>
#include <Utils.hpp>

#include "TracedLife.cuh"


namespace game {
using namespace utils;

Life::Life(int width, int height)
    : _width(width)
    , _height(height)
    , _len(width* height)
    , _prev_device_generation(allocate_dev(_prev_device_generation, _len))
    , _next_device_generation(allocate_dev(_next_device_generation, _len))
{}

Life::~Life() {
    hipFree(_prev_device_generation);
    hipFree(_next_device_generation);
}

ATTRIBS void Life::initialize() {
    static constexpr Cell SPACE[] = {0b1, 0b0, 0b0};
    static constexpr size_t SIZE = sizeof(SPACE) / sizeof(Cell);

    Cell* initial = new Cell[_len];
    for (int i = 0; i < _len; ++i) {
        size_t index = 0 + rand() % SIZE;
        initial[i] = SPACE[index];
    }
    HANDLE_ERROR(hipMemcpy(_prev_device_generation, initial,
                            _len * sizeof(Cell), hipMemcpyHostToDevice));

    delete[] initial;
}

ATTRIBS int Life::eval_index(int w_pos, int h_pos) const {
    if (w_pos == -1) { w_pos = width() - 1; }
    if (w_pos == width()) { w_pos = 0; }
    if (h_pos == -1) { h_pos = height() - 1; }
    if (h_pos == height()) { h_pos = 0; }

    int index = w_pos + (width() * h_pos);
    assert(index < len());
    return index;
}

ATTRIBS Cell Life::cell_status(int w_pos, int h_pos) const {
    const int current_idx = eval_index(w_pos, h_pos);
    assert(current_idx < len());

    Cell neighbours[8];
    neighbours[0] = _prev_device_generation[eval_index(w_pos - 1, h_pos - 1)];
    neighbours[1] = _prev_device_generation[eval_index(w_pos, h_pos - 1)];
    neighbours[2] = _prev_device_generation[eval_index(w_pos + 1, h_pos - 1)];
    neighbours[3] = _prev_device_generation[eval_index(w_pos - 1, h_pos)];
    neighbours[4] = _prev_device_generation[eval_index(w_pos + 1, h_pos)];
    neighbours[5] = _prev_device_generation[eval_index(w_pos - 1, h_pos + 1)];
    neighbours[6] = _prev_device_generation[eval_index(w_pos, h_pos + 1)];
    neighbours[7] = _prev_device_generation[eval_index(w_pos + 1, h_pos + 1)];

    int count = 0;
    for (int i = 0; i < 8; ++i) { count += neighbours[i] & 0x1 == 0x1 ? 1 : 0; }

    const bool alive = _prev_device_generation[current_idx] & 0x1 == 0x1 ?
                       count == 2 || count == 3 : count == 3;
    Cell cell = _prev_device_generation[current_idx];
    cell = cell << 1;
    cell = cell | (alive ? 0b1 : 0b0);
    return cell;
}

ATTRIBS Cell Life::eval_cell(int w_pos, int h_pos) {
    const int current_idx = eval_index(w_pos, h_pos);
    assert(current_idx < len());

    const Cell status = cell_status(w_pos, h_pos);
    _next_device_generation[current_idx] = status;
    return status;
}

ATTRIBS Cell* Life::prev() { return _prev_device_generation; }
ATTRIBS Cell* Life::next() { return _next_device_generation; }

ATTRIBS int Life::len() const { return _len; }
ATTRIBS int Life::width() const { return _width; }
ATTRIBS int Life::height() const { return _height; }


ATTRIBS Color covert_to_color(Cell cell) {
    if (cell & 0x1 == 0x1) { return Color(1.0, 0.0, 0.0); }
    if (cell & 0x2 == 0x2) { return Color(0.0, 0.0, 1.0); }
    return Color(0.0, 0.0, 0.0);
}

__global__ static void __render__(game::Life* ctx, game::Color* frame) {
    const int current_w = blockIdx.x;
    const int current_h = threadIdx.x;
    const int current_i = ctx->eval_index(current_w, current_h);
    assert(current_i < ctx->len());

    game::Color color =
        covert_to_color(ctx->eval_cell(current_w, current_h));

    frame[current_i].r = color.r;
    frame[current_i].g = color.g;
    frame[current_i].b = color.b;
}

__global__ static void __swap__(game::Life* ctx) {
    const int current_w = blockIdx.x;
    const int current_h = threadIdx.x;
    const int current_i = ctx->eval_index(current_w, current_h);
    assert(current_i < ctx->len());

    ctx->prev()[current_i] = ctx->next()[current_i];
}


Game::Game(const Life& life)
    : _host_frame(new Color[life.len()])
    , _dev_frame(utils::allocate_dev(_dev_frame, life.len()))
    , _device_ctx(utils::allocate_managed(_device_ctx, life))
    , _block_width(life.width())
    , _thread_height(life.height())
{}

Game::~Game() {
    delete [] _host_frame;
    hipFree(_dev_frame);
    hipFree(_device_ctx);
}

void Game::render() {
    hipDeviceSynchronize();
    __render__<<<_block_width, _thread_height>>>(_device_ctx, _dev_frame);
    hipDeviceSynchronize();
    __swap__<<<_block_width, _thread_height>>>(_device_ctx);
    hipDeviceSynchronize();
    HANDLE_ERROR(hipMemcpy(_host_frame, _dev_frame,
                            _device_ctx->len() * sizeof(Color),
                            hipMemcpyDeviceToHost));
    glDrawPixels(_device_ctx->width(), _device_ctx->height(), GL_RGB, GL_FLOAT,
                 glm::value_ptr(*_host_frame));
}

}
